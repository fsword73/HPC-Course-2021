#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include<hip/hip_runtime.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
 
// CUDA kernel. One thread execute 1 workload of C = A + B
__global__ void vecAdd(float *a, float *b, float *c, int n){
    int i = blockIdx.x*blockDim.x+threadIdx.x; 
    
    if (i < n)
        c[i] = a[i] * a[i] + b[i] * b[i];
}

#define ARR_SIZE (1024*16) 
int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = ARR_SIZE;
 
    // Host vectors
    float *h_a;
    float *h_b;
    float *h_c;
 
    // Device input vectors
    float *d_a;
    float *d_b;
    float *d_c;
 
    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(float);
 
    // Allocate memory  on host
    h_a = (float*)malloc(bytes);
    h_b = (float*)malloc(bytes);
    h_c = (float*)malloc(bytes);
 
    // Allocate memory  on GPU

    gpuErrchk(hipMalloc(&d_a, bytes));
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    int i;
    // Initialize on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = sinf(i);
        h_b[i] = cosf(i);
    }
 
    // Copy from host to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = 1024;
 
    // total blocks in grid
    gridSize = (int)ceil((float)n/blockSize);
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy result back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
    // Result must be close to 1
    double sum = 0;
    for(i=0; i<n; i++)
        sum += h_c[i];
    printf("final result: %f\n", sum/(double)n);
 
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // free host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}

 
